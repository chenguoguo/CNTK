#include "hip/hip_runtime.h"
//
// Copyright (c) Microsoft. All rights reserved.
// Licensed under the MIT license. See LICENSE.md file in the project root for full license information.
//

#include "stdafx.h"
#include "CuDnnConvolutionEngine.h"
#include "GPUMatrix.h"
#ifdef USE_CUDNN
#include <typeinfo>
#include <typeindex>
#include <hipDNN.h>
#include "CuDnnConvolutionEngine.cuh"

template <>
const char* CudaErrString<hipdnnStatus_t>(hipdnnStatus_t x)
{
    return hipdnnGetErrorString(x);
}

// A note on the formats: CNTK originally used NHWC for input/output tensors and CHWN for filters.
// Such formats have very limited support in cuDNN and not used in other frameworks.
// CNTK with cuDNN by default uses NCHW formats for both inputs/outputs and filters.
#define TENSOR_FORMAT HIPDNN_TENSOR_NCHW
#define FILTER_FORMAT HIPDNN_TENSOR_NCHW
#endif

namespace Microsoft { namespace MSR { namespace CNTK {

#ifdef USE_CUDNN

static bool IsGpu(DEVICEID_TYPE deviceId)
{
    return deviceId >= 0;
}

class CuDnnTensor
{
public:
    CuDnnTensor(const TensorShape& src, hipdnnDataType_t dataType)
        : m_tensor(nullptr)
    {
        CUDNN_CALL(hipdnnCreateTensorDescriptor(&m_tensor));
        // Set cuDNN tesnor dimensions. cuDNN uses row-major format while TensorShape - column major
        // so convertsion is needed.
        const auto& dimsSrc = src.GetDims();
        const auto& stridesSrc = src.GetStrides();
        SmallVector<int> dims(dimsSrc.size() + 1);
        SmallVector<int> strides(stridesSrc.size() + 1);
        assert(dims.size() == strides.size());
        for (int i = 0; i < dimsSrc.size(); i++)
        {
            dims[dims.size() - 1 - i] = (int)dimsSrc[i];
            strides[dims.size() - 1 - i] = (int)stridesSrc[i];
        }
        // Set "minibatch"(aka N) dimension.
        dims[0] = 1;
        strides[0] = strides[1];
        CUDNN_CALL(hipdnnSetTensorNdDescriptor(m_tensor, dataType, (int)src.GetRank() + 1, dims.data(), strides.data()));
    }

    ~CuDnnTensor()
    {
        if (m_tensor != nullptr)
        {
            hipdnnDestroyTensorDescriptor(m_tensor);
            m_tensor = nullptr;
        }
    }

    DISABLE_COPY_AND_MOVE(CuDnnTensor);

private:
    hipdnnTensorDescriptor_t m_tensor;
};

class CuDnnFilter
{
public:
    CuDnnFilter(const ConvolveGeometry& geometry, hipdnnDataType_t dataType)
        : m_filter(nullptr)
    {
        CUDNN_CALL(hipdnnCreateFilterDescriptor(&m_filter));
        // Set cuDNN filter dimensions. cuDNN uses row-major format while TensorShape - column major
        // so convertsion is needed.
        const auto& filt = geometry.KernelShape();
        const auto& maps = geometry.MapCount();
        if (maps.GetRank() > 1 && maps[maps.GetRank() - 1] != maps.GetNumElements())
            InvalidArgument("cuDNN does not support map tensor of this configuration.");
        int mapCount = (int)maps[maps.GetRank() == 1 ? 0 : maps.GetRank() - 1];
        SmallVector<int> dims(filt.GetRank() + 1);
        for (int i = 0; i < filt.GetRank(); i++)
            dims[dims.size() - 1 - i] = (int)filt[i];
        // Set map count(aka K) dimension.
        dims[0] = mapCount;
        CUDNN_CALL(cudnnSetFilterNdDescriptor_v4(m_filter, dataType, FILTER_FORMAT, 
                                                 (int)filt.GetRank() + 1, dims.data()));
    }

    ~CuDnnFilter()
    {
        if (m_filter != nullptr)
        {
            hipdnnDestroyFilterDescriptor(m_filter);
            m_filter = nullptr;
        }
    }

    DISABLE_COPY_AND_MOVE(CuDnnFilter);

private:
    hipdnnFilterDescriptor_t m_filter;
};

class CuDnnConv
{
public:
    CuDnnConv(const ConvolveGeometry& geometry, hipdnnDataType_t dataType)
        : m_conv(nullptr)
    {
        CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&m_conv));
        // Set cuDNN convolution parameters. cuDNN uses row-major format while TensorShape - column major
        // so convertsion is needed.
        SmallVector<int> stride(geometry.InputShape().GetRank());
        SmallVector<int> pad(stride.size());
        for (int i = 0; i < stride.size(); i++)
        {
            stride[stride.size() - 1 - i] = (int)geometry.GetStride(i);
            pad[stride.size() - 1 - i] = geometry.GetLowerPad(i);
        }
        SmallVector<int> upscale(stride.size(), 1);
        CUDNN_CALL(hipdnnSetConvolutionNdDescriptor(m_conv, (int)stride.size(), pad.data(),
                                                   stride.data(), upscale.data(),
                                                   HIPDNN_CROSS_CORRELATION, dataType));
    }

    ~CuDnnConv()
    {
        if (m_conv != nullptr)
        {
            hipdnnDestroyConvolutionDescriptor(m_conv);
            m_conv = nullptr;
        }
    }

    DISABLE_COPY_AND_MOVE(CuDnnConv);

private:
    hipdnnConvolutionDescriptor_t m_conv;
};

template <class ElemType>
class CuDnnConvolutionEngine : public ConvolutionEngine<ElemType>
{
public:
    using Base = ConvolutionEngine<ElemType>;
    using typename Base::Mat;

public:
    CuDnnConvolutionEngine(ConvolveGeometryPtr geometry, DEVICEID_TYPE deviceId, ImageLayoutKind imageLayout, size_t maxTempMemSizeInSamples)
        : Base(geometry, deviceId, imageLayout, maxTempMemSizeInSamples), m_dataType(GetDataType()), 
        m_inT(geometry->InputShape(), m_dataType), m_outT(geometry->OutputShape(), m_dataType),
        m_filter(*geometry, m_dataType), m_conv(*geometry, m_dataType)
    {
        CUDNN_CALL(hipdnnCreate(&m_cudnn));
        CUDNN_CALL(hipdnnSetStream(m_cudnn, GetStream()));
    }

    ~CuDnnConvolutionEngine()
    {
        if (m_cudnn != nullptr)
        {
            hipdnnDestroy(m_cudnn);
            m_cudnn = nullptr;
        }
    }

protected:
    using Base::m_geometry;
    using Base::m_deviceId;
    using Base::m_imageLayout;
    using Base::m_maxTempMemSizeInSamples;

    void EnsureCompatible() override
    {
        if (m_imageLayout != ImageLayoutKind::CHW)
            RuntimeError("cuDNN convolution engine supports only CHW/cudnn layout.");
        if (!IsGpu(m_deviceId))
            RuntimeError("cuDNN convolution engine supports GPU devices only.");
    }

    void ForwardCore(size_t batchSize, const Mat& in, const Mat& filter, Mat& out, Mat& workspace) override
    {
        UNUSED(batchSize); UNUSED(in); UNUSED(filter); UNUSED(out); UNUSED(workspace);

        //// Find best algo and allocate temp buffer, if needed.
        //auto finder = [&](int& calgo, hipdnnConvolutionFwdAlgoPerf_t algoPerf[MaxAlgoCount]) -> hipdnnStatus_t
        //{
        //    return hipdnnFindConvolutionForwardAlgorithm(m_cudnn, t(inT), f(filterT), cd(convDesc), t(outT), MaxAlgoCount, &calgo, algoPerf);
        //};
        //FindBestAlgo(t(inT), m_fwdAlgo, finder);
        //if (m_fwdAlgo.Algo.memory > 0)
        //    workspace.Resize((m_fwdAlgo.Algo.memory + sizeof(ElemType) - 1) / sizeof(ElemType), 1);
        //// Perform forward convolution operation.
        //auto err = hipdnnConvolutionForward(m_cudnn, &C::One, t(inT), ptr(in), f(filterT), ptr(filter), cd(convDesc),
        //                                   m_fwdAlgo.Algo.algo, ptr(workspace), m_fwdAlgo.Algo.memory, &C::Zero, t(outT), ptr(out));
        //// There might be a case where cuDNN fails due to workspace being too small, try using no-workspace algo instead.
        //// REVIEW alexeyk: NVIDIA is currently reviewing this issue.
        //if (HIPDNN_STATUS_INVALID_VALUE == err && m_fwdAlgo.Algo.memory > 0)
        //{
        //    auto err2 = hipdnnConvolutionForward(m_cudnn, &C::One, t(inT), ptr(in), f(filterT), ptr(filter), cd(convDesc),
        //                                        m_fwdAlgo.NoWorkspaceAlgo, nullptr, 0, &C::Zero, t(outT), ptr(out));
        //    // Update original error in case of success.
        //    if (HIPDNN_STATUS_SUCCESS == err2)
        //        err = HIPDNN_STATUS_SUCCESS;
        //}
        //CUDNN_CALL(err);
    }

    void BackwardDataCore(size_t batchSize, const Mat& srcGrad, const Mat& filter, Mat& grad, Mat& workspace) override
    {
        UNUSED(batchSize); UNUSED(srcGrad); UNUSED(filter); UNUSED(grad); UNUSED(workspace);
    }

    void BackwardFilterCore(size_t batchSize, const Mat& srcGrad, const Mat& in, Mat& filter, bool allowReuse, Mat& workspace) override
    {
        UNUSED(batchSize); UNUSED(srcGrad); UNUSED(filter); UNUSED(in); UNUSED(allowReuse); UNUSED(workspace);
    }

private:
    static hipdnnDataType_t GetDataType()
    {
        if (typeid(ElemType) == typeid(float))
            return HIPDNN_DATA_FLOAT;
        else if (typeid(ElemType) == typeid(double))
            return HIPDNN_DATA_DOUBLE;
        else
            InvalidArgument("cuDNN engine currently supports only single and double precision data types.");
    }

private:
    // REVIEW alexeyk: this might be static.
    hipdnnHandle_t m_cudnn;
    hipdnnDataType_t m_dataType;
    CuDnnTensor m_inT;
    CuDnnTensor m_outT;
    CuDnnFilter m_filter;
    CuDnnConv m_conv;
};

template <class ElemType>
std::unique_ptr<ConvolutionEngine<ElemType>> CuDnnConvolutionEngineFactory<ElemType>::CreateConvEngine(
    ConvolveGeometryPtr geometry, DEVICEID_TYPE deviceId, ImageLayoutKind imageLayout, size_t maxTempMemSizeInSamples)
{
    return std::make_unique<CuDnnConvolutionEngine<ElemType>>(geometry, deviceId, imageLayout, maxTempMemSizeInSamples);
}

//class CuDnnTensor4D : public ConvolutionTensor4D
//{
//public:
//    CuDnnTensor4D(size_t w, size_t h, size_t c, size_t n, hipdnnDataType_t dataType)
//        : ConvolutionTensor4D(w, h, c, n), m_dataType(dataType), m_tensor(nullptr)
//    {
//        CUDNN_CALL(hipdnnCreateTensorDescriptor(&m_tensor));
//        CUDNN_CALL(hipdnnSetTensor4dDescriptor(m_tensor, TENSOR_FORMAT, dataType,
//                                              static_cast<int>(n), static_cast<int>(c), static_cast<int>(h), static_cast<int>(w)));
//    }
//
//public:
//    operator hipdnnTensorDescriptor_t() const
//    {
//        return m_tensor;
//    }
//
//    ~CuDnnTensor4D() noexcept
//    {
//        if (m_tensor != nullptr)
//        {
//            hipdnnDestroyTensorDescriptor(m_tensor);
//            m_tensor = nullptr;
//        }
//    }
//
//    void setN(size_t newN) override
//    {
//        ConvolutionTensor4D::setN(newN);
//        CUDNN_CALL(hipdnnSetTensor4dDescriptor(m_tensor, TENSOR_FORMAT, m_dataType,
//                                              static_cast<int>(n()), static_cast<int>(c()), static_cast<int>(h()), static_cast<int>(w())));
//    }
//
//private:
//    hipdnnDataType_t m_dataType;
//    hipdnnTensorDescriptor_t m_tensor;
//};
//
//class CuDnnFilter : public ConvolutionFilter
//{
//public:
//    CuDnnFilter(size_t w, size_t h, size_t c, size_t k, hipdnnDataType_t dataType)
//        : ConvolutionFilter(w, h, c, k), m_filter(nullptr)
//    {
//        CUDNN_CALL(hipdnnCreateFilterDescriptor(&m_filter));
//        CUDNN_CALL(cudnnSetFilter4dDescriptor_v4(m_filter, dataType, FILTER_FORMAT,
//                                                 static_cast<int>(k), static_cast<int>(c), static_cast<int>(h), static_cast<int>(w)));
//    }
//
//public:
//    operator hipdnnFilterDescriptor_t() const
//    {
//        return m_filter;
//    }
//
//    ~CuDnnFilter() noexcept
//    {
//        if (m_filter != nullptr)
//        {
//            hipdnnDestroyFilterDescriptor(m_filter);
//            m_filter = nullptr;
//        }
//    }
//
//private:
//    hipdnnFilterDescriptor_t m_filter;
//};
//
//class CuDnnConvolutionDescriptor : public ConvolutionDescriptor
//{
//public:
//    CuDnnConvolutionDescriptor(size_t wStride, size_t hStride, size_t wPad, size_t hPad)
//        : ConvolutionDescriptor(wStride, hStride, wPad > 0 || hPad > 0), m_conv(nullptr)
//    {
//        CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&m_conv));
//        CUDNN_CALL(hipdnnSetConvolution2dDescriptor(m_conv,
//                                                   static_cast<int>(hPad), static_cast<int>(wPad),
//                                                   static_cast<int>(hStride), static_cast<int>(wStride),
//                                                   1, 1, HIPDNN_CROSS_CORRELATION));
//    }
//
//public:
//    operator hipdnnConvolutionDescriptor_t() const
//    {
//        return m_conv;
//    }
//
//    ~CuDnnConvolutionDescriptor() noexcept
//    {
//        if (m_conv != nullptr)
//        {
//            hipdnnDestroyConvolutionDescriptor(m_conv);
//            m_conv = nullptr;
//        }
//    }
//
//private:
//    hipdnnConvolutionDescriptor_t m_conv;
//};
//
//class CuDnnPoolingDescriptor : public PoolingDescriptor
//{
//public:
//    CuDnnPoolingDescriptor(PoolKind kind, size_t w, size_t h, size_t wStride, size_t hStride, size_t wPad, size_t hPad)
//        : PoolingDescriptor(kind, w, h, wStride, hStride, wPad, hPad), m_pool(nullptr)
//    {
//        assert(kind == PoolKind::Max || kind == PoolKind::Average);
//
//        CUDNN_CALL(hipdnnCreatePoolingDescriptor(&m_pool));
//        CUDNN_CALL(hipdnnSetPooling2dDescriptor(m_pool,
//                                               kind == PoolKind::Max ? HIPDNN_POOLING_MAX : HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING,
//                                               static_cast<int>(h), static_cast<int>(w),
//                                               static_cast<int>(hPad), static_cast<int>(wPad),
//                                               static_cast<int>(hStride), static_cast<int>(wStride)));
//    }
//
//public:
//    operator hipdnnPoolingDescriptor_t() const
//    {
//        return m_pool;
//    }
//
//    ~CuDnnPoolingDescriptor() noexcept
//    {
//        if (m_pool != nullptr)
//        {
//            hipdnnDestroyPoolingDescriptor(m_pool);
//            m_pool = nullptr;
//        }
//    }
//
//private:
//    hipdnnPoolingDescriptor_t m_pool;
//};
//
//template <typename CuDnnT, typename In>
//static CuDnnT& As(In& src)
//{
//    // Do dynamic_cast only in debug builds and static_cast in release builds.
//    assert(dynamic_cast<CuDnnT*>(&src) != nullptr);
//    return static_cast<CuDnnT&>(src);
//}
//static const CuDnnTensor4D& t(const ConvolutionTensor4D& src)
//{
//    return As<const CuDnnTensor4D>(src);
//}
//static const CuDnnFilter& f(const ConvolutionFilter& src)
//{
//    return As<const CuDnnFilter>(src);
//}
//static const CuDnnConvolutionDescriptor& cd(const ConvolutionDescriptor& src)
//{
//    return As<const CuDnnConvolutionDescriptor>(src);
//}
//static const CuDnnPoolingDescriptor& p(const PoolingDescriptor& src)
//{
//    return As<const CuDnnPoolingDescriptor>(src);
//}
//template <typename ElemType>
//static ElemType* ptr(Matrix<ElemType>& src)
//{
//    return src.BufferPointer();
//}
//template <typename ElemType>
//static const ElemType* ptr(const Matrix<ElemType>& src)
//{
//    return src.BufferPointer();
//}
//
//template <typename ElemType>
//struct Consts
//{
//    static const ElemType Zero;
//    static const ElemType One;
//};
//template <>
//const float Consts<float>::One = 1;
//template <>
//const double Consts<double>::One = 1;
//template <>
//const float Consts<float>::Zero = 0;
//template <>
//const double Consts<double>::Zero = 0;
//
//template <typename ElemType>
//class CuDnnConvolutionEngine : public ConvolutionEngine<ElemType>
//{
//public:
//    using Base = ConvolutionEngine<ElemType>;
//    using typename Base::Mat;
//    using typename Base::Tensor4D;
//    using typename Base::Filter;
//    using typename Base::ConvDesc;
//
//    CuDnnConvolutionEngine(DEVICEID_TYPE deviceId, ImageLayoutKind imageLayout, size_t maxTempMemSizeInSamples, BatchNormImpl bnImpl)
//        : Base(deviceId, imageLayout), m_maxTempMemSizeInSamples(maxTempMemSizeInSamples), m_bnImpl(bnImpl), m_stream(GetStream()), m_cudnn(nullptr)
//    {
//        CUDNN_CALL(hipdnnCreate(&m_cudnn));
//        CUDNN_CALL(hipdnnSetStream(m_cudnn, m_stream));
//    }
//
//    ~CuDnnConvolutionEngine()
//    {
//        if (m_cudnn != nullptr)
//        {
//            hipdnnDestroy(m_cudnn);
//            m_cudnn = nullptr;
//        }
//    }
//
//protected:
//    using Base::m_deviceId;
//    using Base::m_imageLayout;
//
//    void EnsureCompatible() override
//    {
//        if (m_imageLayout != ImageLayoutKind::CHW)
//            RuntimeError("cuDNN convolution engine supports only CHW/cudnn layout.");
//        if (!IsGpu(m_deviceId))
//            RuntimeError("cuDNN convolution engine supports GPU devices only.");
//    }
//
//    void ForwardCore(const Tensor4D& inT, const Mat& in, const Filter& filterT, const Mat& filter, const ConvDesc& convDesc,
//                     const Tensor4D& outT, Mat& out, Mat& workspace) override
//    {
//        // Find best algo and allocate temp buffer, if needed.
//        auto finder = [&](int& calgo, hipdnnConvolutionFwdAlgoPerf_t algoPerf[MaxAlgoCount]) -> hipdnnStatus_t
//        {
//            return hipdnnFindConvolutionForwardAlgorithm(m_cudnn, t(inT), f(filterT), cd(convDesc), t(outT), MaxAlgoCount, &calgo, algoPerf);
//        };
//        FindBestAlgo(t(inT), m_fwdAlgo, finder);
//        if (m_fwdAlgo.Algo.memory > 0)
//            workspace.Resize((m_fwdAlgo.Algo.memory + sizeof(ElemType) - 1) / sizeof(ElemType), 1);
//        // Perform forward convolution operation.
//        auto err = hipdnnConvolutionForward(m_cudnn, &C::One, t(inT), ptr(in), f(filterT), ptr(filter), cd(convDesc),
//                                           m_fwdAlgo.Algo.algo, ptr(workspace), m_fwdAlgo.Algo.memory, &C::Zero, t(outT), ptr(out));
//        // There might be a case where cuDNN fails due to workspace being too small, try using no-workspace algo instead.
//        // REVIEW alexeyk: NVIDIA is currently reviewing this issue.
//        if (HIPDNN_STATUS_INVALID_VALUE == err && m_fwdAlgo.Algo.memory > 0)
//        {
//            auto err2 = hipdnnConvolutionForward(m_cudnn, &C::One, t(inT), ptr(in), f(filterT), ptr(filter), cd(convDesc),
//                                                m_fwdAlgo.NoWorkspaceAlgo, nullptr, 0, &C::Zero, t(outT), ptr(out));
//            // Update original error in case of success.
//            if (HIPDNN_STATUS_SUCCESS == err2)
//                err = HIPDNN_STATUS_SUCCESS;
//        }
//        CUDNN_CALL(err);
//    }
//
//    void BackwardDataCore(const Tensor4D& srcGradT, const Mat& srcGrad, const Filter& filterT, const Mat& filter, const ConvDesc& convDesc,
//                          const Tensor4D& gradT, Mat& grad, Mat& workspace) override
//    {
//        // Find best algo and allocate temp buffer, if needed.
//        auto finder = [&](int& calgo, hipdnnConvolutionBwdDataAlgoPerf_t algoPerf[MaxAlgoCount]) -> hipdnnStatus_t
//        {
//            return hipdnnFindConvolutionBackwardDataAlgorithm(m_cudnn, f(filterT), t(srcGradT), cd(convDesc), t(gradT), MaxAlgoCount, &calgo, algoPerf);
//        };
//        FindBestAlgo(t(srcGradT), m_backDataAlgo, finder);
//        if (m_backDataAlgo.Algo.memory > 0)
//            workspace.Resize((m_backDataAlgo.Algo.memory + sizeof(ElemType) - 1) / sizeof(ElemType), 1);
//        // Compute gradients with respect to the output tensor (data).
//        CUDNN_CALL(hipdnnConvolutionBackwardData(m_cudnn, &C::One, f(filterT), ptr(filter), t(srcGradT), ptr(srcGrad), cd(convDesc), m_backDataAlgo.Algo.algo,
//                                                ptr(workspace), m_backDataAlgo.Algo.memory, &C::One, t(gradT), ptr(grad)));
//    }
//
//    void BackwardFilterCore(const Tensor4D& srcGradT, const Mat& srcGrad, const Tensor4D& inT, const Mat& in, const ConvDesc& convDesc,
//                            const Filter& filterT, Mat& filter, bool /*allowReuse*/, Mat& workspace) override
//    {
//        // Find best algo and allocate temp buffer, if needed.
//        auto finder = [&](int& calgo, hipdnnConvolutionBwdFilterAlgoPerf_t algoPerf[MaxAlgoCount]) -> hipdnnStatus_t
//        {
//            return hipdnnFindConvolutionBackwardFilterAlgorithm(m_cudnn, t(inT), t(srcGradT), cd(convDesc), f(filterT), MaxAlgoCount, &calgo, algoPerf);
//        };
//        FindBestAlgo(t(inT), m_backFiltAlgo, finder);
//        if (m_backFiltAlgo.Algo.memory > 0)
//            workspace.Resize((m_backFiltAlgo.Algo.memory + sizeof(ElemType) - 1) / sizeof(ElemType), 1);
//        // Compute gradients with respect to the output tensor (data).
//        CUDNN_CALL(hipdnnConvolutionBackwardFilter(m_cudnn, &C::One, t(inT), ptr(in), t(srcGradT), ptr(srcGrad), cd(convDesc), m_backFiltAlgo.Algo.algo,
//                                                  ptr(workspace), m_backFiltAlgo.Algo.memory, &C::One, f(filterT), ptr(filter)));
//    }
//
//    void EnsureCompatibleBatchNorm(bool spatial) override
//    {
//        if (!IsGpu(m_deviceId))
//            InvalidArgument("cuDNN engine does not support batch normalization on CPUs.");
//        if (spatial && m_imageLayout != ImageLayoutKind::CHW)
//            InvalidArgument("cuDNN engine batch normalization currently supports only CHW data layout for convolutional nodes.");
//    }
//
//    void NormalizeBatchCore(const Tensor4D& inT, const Mat& in, const Tensor4D& scaleBiasT, const Mat& scale, const Mat& bias,
//                            bool spatial, double expAvgFactor, Mat& runMean, Mat& runInvStdDev, Mat& out,
//                            double epsilon, Mat& saveMean, Mat& saveInvStdDev) override
//    {
//        if (m_bnImpl == BatchNormImpl::CuDnn)
//        {
//            hipdnnBatchNormMode_t mode = spatial ? HIPDNN_BATCHNORM_SPATIAL : HIPDNN_BATCHNORM_PER_ACTIVATION;
//            // cuDNN will fail with BAD_PARAM if epsilon < HIPDNN_BN_MIN_EPSILON.
//            epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);
//            CUDNN_CALL(hipdnnBatchNormalizationForwardTraining(m_cudnn, mode, &C::One, &C::Zero, t(inT), ptr(in), t(inT), ptr(out),
//                t(scaleBiasT), ptr(scale), ptr(bias), expAvgFactor, ptr(runMean), ptr(runInvStdDev), 
//                epsilon, ptr(saveMean), ptr(saveInvStdDev)));
//        }
//        else if (m_bnImpl == BatchNormImpl::Cntk)
//        {
//            epsilon = std::max(epsilon, 1e-9);
//            CUDA_CALL(BatchNormalizationForwardTraining(inT, spatial, ptr(in), ptr(out), ptr(scale), ptr(bias),
//                                                        expAvgFactor, ptr(runMean), ptr(runInvStdDev),
//                                                        epsilon, ptr(saveMean), ptr(saveInvStdDev), m_stream));
//        }
//        else
//            RuntimeError("Provided batch norm implementation (%d) is not supported.", m_bnImpl);
//    }
//
//    void NormalizeBatchInferenceCore(const Tensor4D& inT, const Mat& in, const Tensor4D& scaleBiasT, const Mat& scale, const Mat& bias,
//                                     bool spatial, const Mat& runMean, const Mat& runInvStdDev, Mat& out) override
//    {
//        if (m_bnImpl == BatchNormImpl::CuDnn)
//        {
//            hipdnnBatchNormMode_t mode = spatial ? HIPDNN_BATCHNORM_SPATIAL : HIPDNN_BATCHNORM_PER_ACTIVATION;
//            CUDNN_CALL(hipdnnBatchNormalizationForwardInference(m_cudnn, mode, &C::One, &C::Zero, t(inT), ptr(in), t(inT), ptr(out),
//                                                               t(scaleBiasT), ptr(scale), ptr(bias), ptr(runMean), ptr(runInvStdDev), HIPDNN_BN_MIN_EPSILON));
//        }
//        else if (m_bnImpl == BatchNormImpl::Cntk)
//        {
//            CUDA_CALL(BatchNormalizationForwardInference(inT, spatial, ptr(in), ptr(out), ptr(scale), ptr(bias),
//                                                         ptr(runMean), ptr(runInvStdDev), m_stream));
//        }
//        else
//            RuntimeError("Provided batch norm implementation (%d) is not supported.", m_bnImpl);
//    }
//
//    void BackwardNormalizeBatchCore(const Tensor4D& inT, const Mat& in, const Mat& srcGrad, Mat& grad,
//                                    const Tensor4D& scaleBiasT, const Mat& scale, bool spatial, const Mat& saveMean, const Mat& saveInvStdDev,
//                                    Mat& scaleGrad, Mat& biasGrad) override
//    {
//        if (m_bnImpl == BatchNormImpl::CuDnn)
//        {
//            hipdnnBatchNormMode_t mode = spatial ? HIPDNN_BATCHNORM_SPATIAL : HIPDNN_BATCHNORM_PER_ACTIVATION;
//// REVIEW alexeyk: remove once Philly is upgraded to prod version.
//#if CUDNN_PATCHLEVEL >= 7
//            CUDNN_CALL(hipdnnBatchNormalizationBackward(m_cudnn, mode, &C::One, &C::One, &C::One, &C::One, t(inT), ptr(in), t(inT), ptr(srcGrad), t(inT), ptr(grad),
//                                                       t(scaleBiasT), ptr(scale), ptr(scaleGrad), ptr(biasGrad), HIPDNN_BN_MIN_EPSILON, ptr(saveMean), ptr(saveInvStdDev)));
//#else
//            CUDNN_CALL(hipdnnBatchNormalizationBackward(m_cudnn, mode, &C::One, &C::One, t(inT), ptr(in), t(inT), ptr(srcGrad), t(inT), ptr(grad),
//                t(scaleBiasT), ptr(scale), ptr(scaleGrad), ptr(biasGrad), HIPDNN_BN_MIN_EPSILON, ptr(saveMean), ptr(saveInvStdDev)));
//#endif
//
//        }
//        else if (m_bnImpl == BatchNormImpl::Cntk)
//        {
//            CUDA_CALL(BatchNormalizationBackward(inT, spatial, ptr(in), ptr(srcGrad), ptr(grad), ptr(scale), ptr(scaleGrad), ptr(biasGrad),
//                                                 ptr(saveMean), ptr(saveInvStdDev), m_stream));
//        }
//        else
//            RuntimeError("Provided batch norm implementation (%d) is not supported.", m_bnImpl);
//    }
//
//private:
//    static const int MaxAlgoCount = 10;
//
//    template <typename TAlgo, typename TFinder>
//    void FindBestAlgo(const CuDnnTensor4D& t, TAlgo& algo, TFinder finder)
//    {
//        if (!algo.NeedAutotuning(t))
//            return;
//        using CuDnnAlgoT = decltype(TAlgo::Algo);
//        CuDnnAlgoT algoPerf[MaxAlgoCount];
//        int calgo = 0;
//        CUDNN_CALL(finder(calgo, algoPerf));
//        assert(calgo > 0);
//        size_t maxMem = m_maxTempMemSizeInSamples == 0 ? (std::numeric_limits<size_t>::max)() : t.w() * t.h() * t.c() * m_maxTempMemSizeInSamples * sizeof(ElemType);
//        auto res = std::find_if(algoPerf, algoPerf + calgo,
//            [=](const CuDnnAlgoT& cur)
//            {
//                return cur.status == HIPDNN_STATUS_SUCCESS && cur.memory <= maxMem;
//            });
//        if (res == algoPerf + calgo)
//            RuntimeError("cuDNN could not find suitable algorithm for the current convolution configuration.");
//        algo.CurMBSize = t.n();
//        algo.Algo = *res;
//        res = std::find_if(algoPerf, algoPerf + calgo,
//            [](const CuDnnAlgoT& cur)
//            {
//                return cur.status == HIPDNN_STATUS_SUCCESS && cur.memory == 0;
//            });
//        if (res == algoPerf + calgo)
//        {
//            // In theory, this should never happen.
//            RuntimeError("cuDNN could not find no-workspace algorithm for the current convolution configuration.");
//        }
//        else
//            algo.NoWorkspaceAlgo = (*res).algo;
//    }
//
//private:
//    template <typename T>
//    struct ConvAlgoInfo
//    {
//        using CuDnnAlgoT = decltype(T::algo);
//
//        ConvAlgoInfo()
//            : CurMBSize(0)
//        {
//            Algo.status = HIPDNN_STATUS_NOT_INITIALIZED;
//            NoWorkspaceAlgo = (CuDnnAlgoT)-1;
//        }
//        // Current mini-batch size, needed for re-computing statistics in auto-tuner.
//        size_t CurMBSize;
//        T Algo;
//        CuDnnAlgoT NoWorkspaceAlgo;
//
//        bool NeedAutotuning(const CuDnnTensor4D& t)
//        {
//            // Need to re-run auto-tuner in case minibatch size is increased.
//            // If minibatch size is decreased we assume that previously selected algorithm requires less or the same amount of workspace.
//            // This is done to avoid re-running auto-tuner every time in case minibatch size changes frequently (e.g. when distributed reading is enabled).
//            // REVIEW alexeyk: potentially, this might cause some perf issues if better (faster) algo can be selected for a smaller mininbatch.
//            // We also need to reset auto-tuning status at the beginning of each epoch but ComputationNode currently does not provide such notification.
//            // We assume no other dimensions of tensors can change so we don't check it.
//            // REVIEW alexeyk: review once we get response from NVIDIA.
//            return (Algo.status != HIPDNN_STATUS_SUCCESS || t.n() > CurMBSize);
//        }
//    };
//
//    using C = Consts<ElemType>;
//
//    // REVIEW alexeyk: currently limit is set once in ctor though in CNTK it can be, theoretically, changed in runtime.
//    size_t m_maxTempMemSizeInSamples;
//    BatchNormImpl m_bnImpl;
//    hipdnnHandle_t m_cudnn;
//    hipStream_t m_stream;
//    ConvAlgoInfo<hipdnnConvolutionFwdAlgoPerf_t> m_fwdAlgo;
//    ConvAlgoInfo<hipdnnConvolutionBwdDataAlgoPerf_t> m_backDataAlgo;
//    ConvAlgoInfo<hipdnnConvolutionBwdFilterAlgoPerf_t> m_backFiltAlgo;
//};
//
//template <class ElemType>
//class CuDnnPoolingEngine : public PoolingEngine<ElemType>
//{
//public:
//    using Base = PoolingEngine<ElemType>;
//    using typename Base::Tensor4D;
//    using typename Base::PoolDesc;
//    using typename Base::Mat;
//
//public:
//    CuDnnPoolingEngine(DEVICEID_TYPE deviceId, ImageLayoutKind imageLayout)
//        : Base(deviceId, imageLayout), m_cudnn(nullptr)
//    {
//        CUDNN_CALL(hipdnnCreate(&m_cudnn));
//        CUDNN_CALL(hipdnnSetStream(m_cudnn, GetStream()));
//    }
//
//    ~CuDnnPoolingEngine()
//    {
//        if (m_cudnn != nullptr)
//        {
//            hipdnnDestroy(m_cudnn);
//            m_cudnn = nullptr;
//        }
//    }
//
//protected:
//    using Base::m_deviceId;
//    using Base::m_imageLayout;
//
//    void EnsureCompatible() override
//    {
//        if (m_imageLayout != ImageLayoutKind::CHW)
//            RuntimeError("cuDNN pooling engine supports only CHW/cudnn layout.");
//        if (!IsGpu(m_deviceId))
//            RuntimeError("cuDNN pooling engine supports GPU devices only.");
//    }
//
//    void ForwardCore(const Tensor4D& inT, const Mat& in, const PoolDesc& poolDesc, const Tensor4D& outT, Mat& out) override
//    {
//        CUDNN_CALL(hipdnnPoolingForward(m_cudnn, p(poolDesc), &C::One, t(inT), ptr(in), &C::Zero, t(outT), ptr(out)));
//    }
//
//    void BackwardCore(const Tensor4D& outT, const Mat& out, const Mat& srcGrad, const PoolDesc& poolDesc, const Tensor4D& inT, const Mat& in, Mat& grad) override
//    {
//        CUDNN_CALL(hipdnnPoolingBackward(m_cudnn, p(poolDesc), &C::One, t(outT), ptr(out), t(outT), ptr(srcGrad),
//                                        t(inT), ptr(in), &C::One, t(inT), ptr(grad)));
//    }
//
//private:
//    using C = Consts<ElemType>;
//
//    hipdnnHandle_t m_cudnn;
//};
//
//template <class ElemType>
//typename CuDnnConvolutionEngineFactory<ElemType>::Tensor4DPtr CuDnnConvolutionEngineFactory<ElemType>::CreateTensor(size_t w, size_t h, size_t c, size_t n)
//{
//    // REVIEW alexeyk: assert fires in GCC but not in VC++.
//    // static_assert(false, "cuDNN engine currently supports only single and double precision tensors.");
//    RuntimeError("Not implemented.");
//}
//template <>
//typename CuDnnConvolutionEngineFactory<float>::Tensor4DPtr CuDnnConvolutionEngineFactory<float>::CreateTensor(size_t w, size_t h, size_t c, size_t n)
//{
//    return std::make_unique<CuDnnTensor4D>(w, h, c, n, HIPDNN_DATA_FLOAT);
//}
//template <>
//typename CuDnnConvolutionEngineFactory<double>::Tensor4DPtr CuDnnConvolutionEngineFactory<double>::CreateTensor(size_t w, size_t h, size_t c, size_t n)
//{
//    return std::make_unique<CuDnnTensor4D>(w, h, c, n, HIPDNN_DATA_DOUBLE);
//}
//
//template <class ElemType>
//typename CuDnnConvolutionEngineFactory<ElemType>::FilterPtr CuDnnConvolutionEngineFactory<ElemType>::CreateFilter(size_t w, size_t h, size_t c, size_t k)
//{
//    // REVIEW alexeyk: assert fires in GCC but not in VC++.
//    // static_assert(false, "cuDNN engine currently supports only single and double precision filters.");
//    RuntimeError("Not implemented.");
//}
//template <>
//typename CuDnnConvolutionEngineFactory<float>::FilterPtr CuDnnConvolutionEngineFactory<float>::CreateFilter(size_t w, size_t h, size_t c, size_t k)
//{
//    return std::make_unique<CuDnnFilter>(w, h, c, k, HIPDNN_DATA_FLOAT);
//}
//template <>
//typename CuDnnConvolutionEngineFactory<double>::FilterPtr CuDnnConvolutionEngineFactory<double>::CreateFilter(size_t w, size_t h, size_t c, size_t k)
//{
//    return std::make_unique<CuDnnFilter>(w, h, c, k, HIPDNN_DATA_DOUBLE);
//}
//
//template <class ElemType>
//typename CuDnnConvolutionEngineFactory<ElemType>::ConvDescPtr CuDnnConvolutionEngineFactory<ElemType>::CreateConvDescriptor(
//    const Tensor4D& /*inT*/, const Filter& filterT, size_t wStride, size_t hStride, bool padding)
//{
//    size_t wPad = padding ? filterT.w() / 2 : 0;
//    size_t hPad = padding ? filterT.h() / 2 : 0;
//    return std::make_unique<CuDnnConvolutionDescriptor>(wStride, hStride, wPad, hPad);
//}
//
//template <class ElemType>
//typename CuDnnConvolutionEngineFactory<ElemType>::PoolDescPtr CuDnnConvolutionEngineFactory<ElemType>::CreatePoolDescriptor(
//    typename PoolDesc::PoolKind kind, size_t w, size_t h, size_t wStride, size_t hStride, size_t wPad, size_t hPad)
//{
//    return std::make_unique<CuDnnPoolingDescriptor>(kind, w, h, wStride, hStride, wPad, hPad);
//}
//
//template <class ElemType>
//typename CuDnnConvolutionEngineFactory<ElemType>::ConvEnginePtr CuDnnConvolutionEngineFactory<ElemType>::CreateConvEngine(
//    DEVICEID_TYPE deviceId, ImageLayoutKind imageLayout, size_t maxTempMemSizeInSamples, BatchNormImpl bnImpl)
//{
//    return std::make_unique<CuDnnConvolutionEngine<ElemType>>(deviceId, imageLayout, maxTempMemSizeInSamples, bnImpl);
//}
//
//template <class ElemType>
//typename CuDnnConvolutionEngineFactory<ElemType>::PoolEnginePtr CuDnnConvolutionEngineFactory<ElemType>::CreatePoolEngine(
//    DEVICEID_TYPE deviceId, ImageLayoutKind imageLayout)
//{
//    return std::make_unique<CuDnnPoolingEngine<ElemType>>(deviceId, imageLayout);
//}
//
#else

template <class ElemType>
typename CuDnnConvolutionEngineFactory<ElemType>::Tensor4DPtr CuDnnConvolutionEngineFactory<ElemType>::CreateTensor(size_t, size_t, size_t, size_t)
{
    RuntimeError("The code is compiled without USE_CUDNN macro.");
}

template <class ElemType>
typename CuDnnConvolutionEngineFactory<ElemType>::FilterPtr CuDnnConvolutionEngineFactory<ElemType>::CreateFilter(size_t, size_t, size_t, size_t)
{
    RuntimeError("The code is compiled without USE_CUDNN macro.");
}

template <class ElemType>
typename CuDnnConvolutionEngineFactory<ElemType>::ConvDescPtr CuDnnConvolutionEngineFactory<ElemType>::CreateConvDescriptor(
    const Tensor4D&, const Filter&, size_t, size_t, bool)
{
    RuntimeError("The code is compiled without USE_CUDNN macro.");
}

template <class ElemType>
typename CuDnnConvolutionEngineFactory<ElemType>::PoolDescPtr CuDnnConvolutionEngineFactory<ElemType>::CreatePoolDescriptor(
    typename PoolDesc::PoolKind, size_t, size_t, size_t, size_t, size_t, size_t)
{
    RuntimeError("The code is compiled without USE_CUDNN macro.");
}

template <class ElemType>
typename CuDnnConvolutionEngineFactory<ElemType>::ConvEnginePtr CuDnnConvolutionEngineFactory<ElemType>::CreateConvEngine(DEVICEID_TYPE, ImageLayoutKind, size_t, BatchNormImpl)
{
    RuntimeError("The code is compiled without USE_CUDNN macro.");
}

template <class ElemType>
typename CuDnnConvolutionEngineFactory<ElemType>::PoolEnginePtr CuDnnConvolutionEngineFactory<ElemType>::CreatePoolEngine(DEVICEID_TYPE, ImageLayoutKind)
{
    RuntimeError("The code is compiled without USE_CUDNN macro.");
}

#endif

// REVIEW alexeyk: remove #ifdef once cuDNN becomes mandatory dependency.
#ifdef USE_CUDNN
template <class ElemType>
bool CuDnnConvolutionEngineFactory<ElemType>::IsSupported(DEVICEID_TYPE deviceId)
{
    hipDeviceProp_t props = {0};
    return hipGetDeviceProperties(&props, deviceId) == hipSuccess && props.major >= 3;
#else
    UNUSED(deviceId);
    return false;
#endif
}

template class CuDnnConvolutionEngineFactory<float>;
template class CuDnnConvolutionEngineFactory<double>;

CudaTimer::~CudaTimer()
{
    if (m_start != nullptr)
        CUDA_CALL(hipEventDestroy(reinterpret_cast<hipEvent_t>(m_start)));
    if (m_stop != nullptr)
        CUDA_CALL(hipEventDestroy(reinterpret_cast<hipEvent_t>(m_stop)));
}
void CudaTimer::Start()
{
    hipEvent_t start;
    hipEvent_t stop;
    if (m_start != nullptr)
        CUDA_CALL(hipEventDestroy(reinterpret_cast<hipEvent_t>(m_start)));
    if (m_stop != nullptr)
        CUDA_CALL(hipEventDestroy(reinterpret_cast<hipEvent_t>(m_stop)));
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));
    m_start = start;
    m_stop = stop;
    CUDA_CALL(hipEventRecord(start, GetStream()));
}
void CudaTimer::Stop()
{
    CUDA_CALL(hipEventRecord(reinterpret_cast<hipEvent_t>(m_stop), GetStream()));
    CUDA_CALL(hipEventSynchronize(reinterpret_cast<hipEvent_t>(m_stop)));
}
float CudaTimer::Elapsed()
{
    float ms;
    CUDA_CALL(hipEventElapsedTime(&ms, reinterpret_cast<hipEvent_t>(m_start), reinterpret_cast<hipEvent_t>(m_stop)));
    return ms;
}

} } }
